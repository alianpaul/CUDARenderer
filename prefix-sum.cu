#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

#include "helper.cuh"

__global__ void preScan(unsigned int* deviceInput, unsigned int* deviceOutput, int cnt,
						unsigned int* deviceSum, bool isSum)
{
	extern __shared__ unsigned int temp[];
	int cntInB = blockDim.x * 2;
	int idxInG = cntInB * blockIdx.x + threadIdx.x;

	int idxInB = threadIdx.x;
	temp[2 * idxInB]		= 0;
	temp[2 * idxInB +1]		= 0;

	if (idxInG < cnt)
	{
		temp[idxInB] = deviceInput[idxInG];
	}
	
	if (idxInG + blockDim.x < cnt)
	{
		temp[idxInB + blockDim.x] = deviceInput[idxInG + blockDim.x];
	}

	int offset = 1;
	for (int d = cntInB >> 1; d > 0; d>>=1)
	{
		__syncthreads();
		if (threadIdx.x < d)
		{
			int ai = offset - 1 + offset * (threadIdx.x * 2);
			int bi = ai + offset;
			temp[bi] += temp[ai];
		}
		offset *= 2;
	}

	//before clear the last element, move the last element to deviceSums.
	if (threadIdx.x == 0)
	{
		if (isSum)
		{
			deviceSum[blockIdx.x] = temp[cntInB - 1];
		}
		temp[cntInB - 1] = 0;
	}
	

	//downsweep
	for (int d = 1; d < cntInB; d *=2)
	{
		offset >>= 1;
		__syncthreads();

		if (threadIdx.x < d)
		{

			int ai = offset - 1 + offset * (threadIdx.x * 2);
			int bi = ai + offset;
			unsigned int be = temp[bi];
			temp[bi] += temp[ai];
			temp[ai] = be;
		}
	}

	__syncthreads();
	if (idxInG < cnt)
	{
		deviceOutput[idxInG] = temp[idxInB];
	}

	if (idxInG + blockDim.x < cnt)
	{
		deviceOutput[idxInG + blockDim.x] = temp[idxInB + blockDim.x];
	}
}

__global__ void addInc(unsigned int* deviceInput, unsigned int* deviceOutput, int eleCnt,
					   unsigned int* deviceInc)
{
	__shared__ int inc;
	if (threadIdx.x == 0)
	{
		inc = deviceInc[blockIdx.x];
	}
	__syncthreads();

	int cntInB = blockDim.x * 2;
	int idxInG = blockIdx.x * cntInB + threadIdx.x;

	if (idxInG < eleCnt)
	{
		deviceOutput[idxInG] = deviceInput[idxInG] + inc;
	}

	if (idxInG + blockDim.x < eleCnt)
	{
		deviceOutput[idxInG + blockDim.x] = deviceInput[idxInG + blockDim.x] + inc;
	}

}


/*input:	allocated and initialized device memory
* output:	allocated device memory
* cnt:		size
*/
void prefixSum(unsigned int* deviceInput, unsigned int* deviceOutput, int eleCnt)
{

	/*Test:	
	int eleCnt = 1025;
	unsigned int* deviceInput;
	hipMalloc(&deviceInput, sizeof(unsigned int) * eleCnt);
	unsigned int* deviceOutput;
	hipMalloc(&deviceOutput, sizeof(unsigned int) * eleCnt);

	unsigned int* hostInput;
	hostInput = (unsigned int*)malloc(sizeof(unsigned int) * eleCnt);
	for (size_t i = 0; i < eleCnt; ++i)
	{
		hostInput[i] = 1;
	}
	hipMemcpy(deviceInput, hostInput, sizeof(unsigned int) * eleCnt, hipMemcpyHostToDevice);
	*/

	dim3 blockDim(256);
	int eleCntInB = blockDim.x * 2;
	unsigned int sharedMemSize = eleCntInB * sizeof(unsigned int);

	dim3 gridDim((eleCnt+ eleCntInB - 1) / eleCntInB);
	int blockCnt = gridDim.x;

	unsigned int* deviceSum;
	hipMalloc(&deviceSum, sizeof(unsigned int)*blockCnt);
	unsigned int* deviceInc;
	hipMalloc(&deviceInc, sizeof(unsigned int)*blockCnt);

	preScan<<<gridDim, blockDim, sharedMemSize>>>(deviceInput, deviceOutput, eleCnt,
												  deviceSum, true);
	preScan<<<1, blockDim, sharedMemSize>>>(deviceSum, deviceInc, blockCnt,
											0, false);
	addInc<<<gridDim, blockDim>>>(deviceOutput, deviceOutput, eleCnt,
								  deviceInc);
	
	/*Test Output:
	unsigned int* hostScanOut = (unsigned int*)malloc(sizeof(unsigned int) * eleCnt);
	hipMemcpy(hostScanOut, deviceOutput, sizeof(unsigned int) * eleCnt, hipMemcpyDeviceToHost);
	printf("Final result\n");
	for (size_t i = 0; i < eleCnt; ++i)
	{
		printf("%d ", hostScanOut[i]);
	}
	*/
	hipFree(deviceInc);
	hipFree(deviceSum);
}